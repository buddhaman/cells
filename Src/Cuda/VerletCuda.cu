#include "hip/hip_runtime.h"
#include "VerletCuda.h"
#include <>

// Vector operators for CUDA float2
__device__ __forceinline__ float2 
operator+(float2 a, float2 b) 
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ __forceinline__ float2 
operator-(float2 a, float2 b) 
{
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ __forceinline__ float2 
operator*(float2 a, float b) 
{
    return make_float2(a.x * b, a.y * b);
}

__device__ __forceinline__ float2 
operator*(float b, float2 a) 
{
    return make_float2(a.x * b, a.y * b);
}

__device__ __forceinline__ void 
operator+=(float2& a, float2 b) 
{
    a.x += b.x;
    a.y += b.y;
}

__device__ __forceinline__ void 
operator-=(float2& a, float2 b) 
{
    a.x -= b.x;
    a.y -= b.y;
}

__device__ float 
Length(float2 v) 
{
    return sqrtf(v.x * v.x + v.y * v.y);
}

// Helper functions for vector operations
__device__ float2 normalize(float2 v) {
    float len = Length(v);
    if (len > 0.0001f) {
        return make_float2(v.x / len, v.y / len);
    }
    return make_float2(0.0f, 0.0f);
}

// CUDA kernel for Verlet integration
__global__ void UpdatePositionsKernel(CudaWorld* cuda_world) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Array<VerletParticle> particles = cuda_world->particles;
    if (idx >= particles.size || particles.data[idx].is_static) 
        return;

    VerletParticle* p = &particles.data[idx];
    
    // Store current position
    float2 temp = p->position;
    
    float2 velocity = (p->position - p->old_position) * 0.94f;
    p->position = p->position + velocity;
    p->old_position = temp;
}

__global__ void UpdateConstraintsKernel(CudaWorld* cuda_world) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Array<VerletConstraint> constraints = cuda_world->constraints;
    if (idx >= constraints.size)
        return;

    VerletConstraint* c = &constraints.data[idx];
    VerletParticle* p1 = c->particle1;
    VerletParticle* p2 = c->particle2;

    // Skip if both particles are static
    if (p1->is_static && p2->is_static)
        return;

    float2 delta = p2->position - p1->position;
    float dist = Length(delta);
    
    // Avoid division by zero
    if (dist < 0.0001f)
        return;
        
    //
    // static inline void
    // VerletSolve(Verlet3Constraint* constraint)
    // {
    //     Vec3 delta = constraint->v1->pos - constraint->v0->pos;
    //     float distance = V3Len(delta);

    //     if (distance == 0.0f) return;

    //     float correction = (distance - constraint->r) / distance;
    //     Vec3 correction_vector = delta * 0.5f * correction;
    //     constraint->v0->pos += correction_vector; 
    //     constraint->v1->pos -= correction_vector;
    // }
    // Calculate the error as a ratio of current length to rest length
    float error = (dist - c->rest_length) / (dist);
    float2 correction = delta * error * c->stiffness * 0.5f;

    if (!p1->is_static) p1->position += correction;
    if (!p2->is_static) p2->position -= correction;
}

void UpdateVerletParticles(CudaWorld* cuda_world) 
{
    const int block_size = 256;
    const int num_blocks = (cuda_world->particles.size + block_size - 1) / block_size;
    
    // Update positions with fixed timestep
    UpdatePositionsKernel<<<num_blocks, block_size>>>(cuda_world);
    hipDeviceSynchronize();

    // Solve constraints multiple times for stability
    const int solver_iterations = 10; // Increase for more stability, decrease for performance
    const int constraint_block_size = 256;
    const int num_constraint_blocks = (cuda_world->constraints.size + constraint_block_size - 1) / constraint_block_size;
    
    for (int i = 0; i < solver_iterations; i++) {
        UpdateConstraintsKernel<<<num_constraint_blocks, constraint_block_size>>>(cuda_world);
        hipDeviceSynchronize();
    }
}
